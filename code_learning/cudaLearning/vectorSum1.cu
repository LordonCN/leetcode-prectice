#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>

#include <ctime>

// 初步实现功能 并进行测试

using namespace std;
// 对向量的运算
__global__ void addkernel(int *c,const int *a,const int *b)
{
    int i = threadIdx.x;

// 默认情况下i是从0开始进行调用哎哎的,所以可以用这种方法进行计算
// 只有编译能力2.0以上的显卡才可以调用printf
// 这里不建议使用printf 调试的话应该在cpu上进行输出显示
// printf("now i is %d in thread.\n",i); //该输出可以查看调用的线程序号

    c[i] = a[i] + b[i];

}

int main()
{
    const int arraySize = 20;

    const int a[arraySize] = {1,2,3,4,5,1,2,3,4,5,1,2,3,4,5,1,2,3,4,5};
    const int b[arraySize] = {10,20,30,40,50,10,20,30,40,50,10,20,30,40,50,10,20,30,40,50};
    int c[arraySize] = {0};

// 调用进程块与线程总数为 gd*dt
    int gd = 1;
    int dt = arraySize;

// 测试线程数对计算速度的影响
    clock_t startTime,endTime;
    startTime = clock();
    int *dev_a,*dev_b,*dev_c;

// cuda状态监测
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if(cudaStatus == hipSuccess)
        cout<<"setDevice successful"<<endl;
    else
    {
        cout<<"setDevice failed!"<<endl;
        goto ERROR;
    }


// 创建cuda内存
    cudaStatus = hipMalloc((void**)&dev_c, arraySize * sizeof(int));
    cudaStatus = hipMalloc((void**)&dev_a, arraySize * sizeof(int));
    cudaStatus = hipMalloc((void**)&dev_b, arraySize * sizeof(int));
    if (cudaStatus == hipSuccess){
        cout<<"hipMalloc successful"<<endl;
    }
    else
    {
        cout<<"setDevice failed!"<<endl;
        goto ERROR;
    }


// 将ab传到显卡上
    cudaStatus = hipMemcpy(dev_a,a,arraySize * sizeof(int),hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(dev_b,b,arraySize * sizeof(int),hipMemcpyHostToDevice);
    if (cudaStatus == hipSuccess){
        cout<<"cudacpy successful"<<endl;
    }
    else
    {
        cout<<"cudacpy failed"<<endl;
        goto ERROR;
    }


// 调用kernel函数 网格中线程块为1 线程块中包括5个线程
// 所有线程调用结束后返回
    addkernel<<<gd,dt>>>(dev_c,dev_a,dev_b);


// 这个地方是由设备c传回cpu-c deviceToHost 第一次在这里卡了好久
    cudaStatus = hipMemcpy(c,dev_c,arraySize * sizeof(int),hipMemcpyDeviceToHost);
    if (cudaStatus == hipSuccess){
        cout<<"cudacpy successful"<<endl;
    }
    else
    {
        cout<<"cudacpy failed"<<endl;
        goto ERROR;
    }

    endTime = clock();


    ERROR:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    cout<<"run time is:"<<endTime-startTime<<" And result is:"
        <<c[0]<<" "<<c[1]<<" "<<c[2]<<" "<<c[3]<<" "<<c[4]
        <<c[5]<<" "<<c[6]<<" "<<c[7]<<" "<<c[8]<<" "<<c[9]
        <<c[10]<<" "<<c[11]<<" "<<c[12]<<" "<<c[13]<<" "<<c[14]
        <<endl;


    return 0;
}