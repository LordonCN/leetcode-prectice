//
// Created by Mac_OS_Dong on 2021/3/9.
//

// 显示显卡信息
void showTheDetailOftheMechine()
{
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Num: %d\n", i);
        printf("Device name: %s\n", prop.name);
        printf("Device SM Num: %d\n", prop.multiProcessorCount);
        printf("Share Mem Per Block: %.2fKB\n", prop.sharedMemPerBlock / 1024.0);
        printf("Max Thread Per Block: %d\n", prop.maxThreadsPerBlock);
        printf("Memory Clock Rate (KHz): %d\n",
               prop.memoryClockRate);
        printf("Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf("Peak Memory Bandwidth (GB/s): %.2f\n\n",
               2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
    }
    return;
}


int main() {

    showTheDetailOftheMechine();

    return 0;
}