#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>

#include <time.h>
#include <sys/time.h>

// 参考stackoverflow大佬 添加kernel_wapper function and timecal

using namespace std;
// 对向量的运算
__global__ void addkernel(int *c,const int *a,const int *b)
{
    register int i = threadIdx.x;

// 默认情况下i是从0开始进行调用哎哎的,所以可以用这种方法进行计算
// 只有编译能力2.0以上的显卡才可以调用printf
// 这里不建议使用printf 调试的话应该在cpu上进行输出显示
// printf("now i is %d in thread.\n",i); //该输出可以查看调用的线程序号

    c[i] = a[i] + b[i];

}


void kernel_wapper(int*dev_a,int *dev_b,int *dev_c,const int a[],const int b[],int c[],int arraySize)
{
    hipError_t cudaStatus;
// 创建cuda内存
    cudaStatus = hipMalloc((void**)&dev_c, arraySize * sizeof(int));
    cudaStatus = hipMalloc((void**)&dev_a, arraySize * sizeof(int));
    cudaStatus = hipMalloc((void**)&dev_b, arraySize * sizeof(int));
    if (cudaStatus == hipSuccess){
        cout<<"hipMalloc successful"<<endl;
    }
    else{
        cout<<"setDevice failed!"<<endl;
    }

// 将ab传到显卡上
    cudaStatus = hipMemcpy(dev_a,a,arraySize * sizeof(int),hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(dev_b,b,arraySize * sizeof(int),hipMemcpyHostToDevice);
    if (cudaStatus == hipSuccess){
        cout<<"cudacpy successful"<<endl;
    }
    else
        cout<<"cudacpy failed"<<endl;

// 调用kernel函数 网格中线程块为1 线程块中包括5个线程
// 所有线程调用结束后返回
    addkernel<<<1,arraySize>>>(dev_c,dev_a,dev_b);

// 这个地方是由设备c传回cpu-c deviceToHost 第一次在这里卡了好久
    cudaStatus = hipMemcpy(c,dev_c,arraySize * sizeof(int),hipMemcpyDeviceToHost);
    if (cudaStatus == hipSuccess){
        cout<<"cudacpy successful"<<endl;
    }
    else
        cout<<"cudacpy failed"<<endl;

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

}

int main()
{
    const int arraySize = 20;

    const int a[arraySize] = {1,2,3,4,5,1,2,3,4,5,1,2,3,4,5,1,2,3,4,5};
    const int b[arraySize] = {10,20,30,40,50,10,20,30,40,50,10,20,30,40,50,10,20,30,40,50};
    int c[arraySize] = {0};

// 调用进程块与线程总数为 gd*dt
    int gd = 1;
    int dt = arraySize;

    int *dev_a,*dev_b,*dev_c;

// 运行时间统计
    timeval t1,t2;
    unsigned long et;

// cuda状态监测
    hipError_t cudaStatus;

// 显卡状态监测
    cudaStatus = hipSetDevice(0);
    if(cudaStatus == hipSuccess)
        cout<<"setDevice successful"<<endl;
    else
        cout<<"setDevice failed!"<<endl;

// 计时
    gettimeofday(&t1,NULL);

// kernel任务
    if(cudaStatus == hipSuccess)
        kernel_wapper(dev_a,dev_b,dev_c,a,b,c,dt);

    gettimeofday(&t2,NULL);

// 输出时间
    et = ((t2.tv_sec * 1000000)+t2.tv_usec) - ((t1.tv_sec * 1000000) + t1.tv_usec);
    printf("time 1 = %ld us\n", et);



    cout<<"Result is:"
        <<c[0]<<" "<<c[1]<<" "<<c[2]<<" "<<c[3]<<" "<<c[4]
        <<c[5]<<" "<<c[6]<<" "<<c[7]<<" "<<c[8]<<" "<<c[9]
        <<c[10]<<" "<<c[11]<<" "<<c[12]<<" "<<c[13]<<" "<<c[14]
        <<endl;


    return 0;
}
